#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernels.cuh"

__global__ void stridingMemcpyKernel(unsigned int totalThreadCount, unsigned long long loopCount, uint4* dst, uint4* src, size_t chunkSizeInElement) {
    unsigned long long from = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned long long bigChunkSizeInElement = chunkSizeInElement / 12;
    dst += from;
    src += from;
    uint4* dstBigEnd = dst + (bigChunkSizeInElement * 12) * totalThreadCount;
    uint4* dstEnd = dst + chunkSizeInElement * totalThreadCount;

    for (unsigned int i = 0; i < loopCount; i++) {
        uint4* cdst = dst;
        uint4* csrc = src;

        while (cdst < dstBigEnd) {
            uint4 pipe_0 = *csrc; csrc += totalThreadCount;
            uint4 pipe_1 = *csrc; csrc += totalThreadCount;
            uint4 pipe_2 = *csrc; csrc += totalThreadCount;
            uint4 pipe_3 = *csrc; csrc += totalThreadCount;
            uint4 pipe_4 = *csrc; csrc += totalThreadCount;
            uint4 pipe_5 = *csrc; csrc += totalThreadCount;
            uint4 pipe_6 = *csrc; csrc += totalThreadCount;
            uint4 pipe_7 = *csrc; csrc += totalThreadCount;
            uint4 pipe_8 = *csrc; csrc += totalThreadCount;
            uint4 pipe_9 = *csrc; csrc += totalThreadCount;
            uint4 pipe_10 = *csrc; csrc += totalThreadCount;
            uint4 pipe_11 = *csrc; csrc += totalThreadCount;

            *cdst = pipe_0; cdst += totalThreadCount;
            *cdst = pipe_1; cdst += totalThreadCount;
            *cdst = pipe_2; cdst += totalThreadCount;
            *cdst = pipe_3; cdst += totalThreadCount;
            *cdst = pipe_4; cdst += totalThreadCount;
            *cdst = pipe_5; cdst += totalThreadCount;
            *cdst = pipe_6; cdst += totalThreadCount;
            *cdst = pipe_7; cdst += totalThreadCount;
            *cdst = pipe_8; cdst += totalThreadCount;
            *cdst = pipe_9; cdst += totalThreadCount;
            *cdst = pipe_10; cdst += totalThreadCount;
            *cdst = pipe_11; cdst += totalThreadCount;
        }

        while (cdst < dstEnd) {
            *cdst = *csrc; cdst += totalThreadCount; csrc += totalThreadCount;
        }
    }
}

size_t copyKernel(hipDeviceptr_t dstBuffer, hipDeviceptr_t srcBuffer, size_t size, hipStream_t stream, unsigned long long loopCount) {
    hipDevice_t dev;
    hipCtx_t ctx;

    CU_ASSERT(cuStreamGetCtx(stream, &ctx));
    CU_ASSERT(hipCtxGetDevice(&dev));

    int numSm;
    CU_ASSERT(hipDeviceGetAttribute(&numSm, hipDeviceAttributeMultiprocessorCount, dev));
    unsigned int totalThreadCount = numSm * numThreadPerBlock;

    // adjust size to elements (size is multiple of MB, so no truncation here)
    size_t sizeInElement = size / sizeof(uint4);
    // this truncates the copy
    sizeInElement = totalThreadCount * (sizeInElement / totalThreadCount);

    size_t chunkSizeInElement = sizeInElement / totalThreadCount;

    dim3 gridDim(numSm, 1, 1);
    dim3 blockDim(numThreadPerBlock, 1, 1);
    stridingMemcpyKernel<<<gridDim, blockDim, 0, stream>>> (totalThreadCount, loopCount, (uint4 *)dstBuffer, (uint4 *)srcBuffer, chunkSizeInElement);

    return sizeInElement * sizeof(uint4);
}

__global__ void spinKernel(volatile int *latch, const unsigned long long timeoutClocks)
{
    register unsigned long long endTime = clock64() + timeoutClocks;
    while (!*latch) {
        if (timeoutClocks != ~0ULL && clock64() > endTime) {
            break;
        }
    }
}

hipError_t spinKernel(volatile int *latch, hipStream_t stream, unsigned long long timeoutNs)
{
    int clocksPerMs = 0;
    hipCtx_t ctx;
    hipDevice_t dev;

    CU_ASSERT(cuStreamGetCtx(stream, &ctx));
    CU_ASSERT(hipCtxGetDevice(&dev));

    CU_ASSERT(hipDeviceGetAttribute(&clocksPerMs, hipDeviceAttributeClockRate, dev));

    unsigned long long timeoutClocks = (clocksPerMs * timeoutNs) / 1000;

    spinKernel<<<1, 1, 0, stream>>>(latch, timeoutClocks);

    return hipSuccess;
}
